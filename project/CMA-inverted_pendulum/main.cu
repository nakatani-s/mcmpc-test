#include "hip/hip_runtime.h"
/*
 *  This file is part of MCMPC toolkit.
 *  
 *  MCMPC toolkit -- A toolkit for Model Predictive Control
*/
/* 
    [file]      project/example_inverted_pendulum/main.cu
    [author]    Shintaro Nakatani
    [date]      2022.6.15
*/

#include "../../include/mcmpc_toolkit.cuh"

int main(int argc, char **argv)
{
    time_t time_value;
    struct tm *time_object;
    int before_tm_min;
    time(&time_value);
    time_object = localtime( &time_value );
    before_tm_min = time_object->tm_min;
    
    // mcmpc myMPC;
    // sample_based_newton_method myMPC;
    cma_mpc myMPC;
    // savitzky_golay_filter sgFilter;
    // Optional settings
    // myMPC.Set(HYPERBOLIC, SET_COOLING_METHOD);
    myMPC.Set(TIME_INVARIANT , SET_REFERENCE_TYPE);
    // myMPC.Set(GOLDEN_SECTION, SET_STEP_WIDTH_ADJUSTING_METHOD);
    // myMPC.Set(EIGEN_VALUE_DECOM, SET_SOLVER);
    // 
    float p[2] = {0.024f, 0.2f};
    float Jp = p[0] * pow(p[1], 2) / 3.0f;
    float wall_position = atof(argv[1]) / 100;
    printf("%f th wall position == %f\n", atof(argv[1]),  wall_position);

    float state[OCP_SETTINGS::DIM_OF_STATE] = {0.0f, 0.0f, M_PI+0.03f, 0.0f};
    float u[OCP_SETTINGS::DIM_OF_INPUT] = {0.0f};
    // float param[OCP_SETTINGS::DIM_OF_PARAMETER] = {0.1f, 0.024f, 0.2f, Jp, 1.265f, 1e-6, 9.80665f, -0.25, 0.1, 0.46}; // <== Success 2022.9.1
    float param[OCP_SETTINGS::DIM_OF_PARAMETER] = {0.1f, 0.024f, 0.2f, Jp, 1.265f, 1e-6, 9.80665f, -0.04, wall_position, 0.55};
    // float constraint[OCP_SETTINGS::DIM_OF_CONSTRAINTS] = {-1.0, 1.0, -0.5, 0.5}; // For utlizing collision
    float constraint[OCP_SETTINGS::DIM_OF_CONSTRAINTS] = {-1.0, 1.0, -0.04, wall_position}; // For predict as constraint 
    float weight_matrix[OCP_SETTINGS::DIM_OF_WEIGHT_MATRIX] = {5.0f, 0.04f, 10.0f, 0.05f, 1.0f};
    float reference[OCP_SETTINGS::DIM_OF_REFERENCE] = { };

    myMPC.Set(state, SET_STATE);
    myMPC.Set(u, SET_INPUT);
    myMPC.Set(param, SET_PARAMETER);
    myMPC.Set(constraint, SET_CONSTRAINT);
    myMPC.Set(weight_matrix, SET_WEIGHT_MATRIX);
    myMPC.Set(reference, SET_REFERENCE);

    for(int t = 0; t < OCP_SETTINGS::SIMULATION_STEPS; t++)
    {
        myMPC.ExecuteMPC( u );

        // sgFilter.Smoothing(u, myMPC.mcmpc_input_sequences);
        // if(t < 10) u[0] = constraint[1];
        myMPC.ExecuteForwardSimulation(state, u, RUNGE_KUTTA_45);

        // if(state[0] >= param[8])
        // {
        //     float collide[3] = {};
        //     float coefficient = param[9];
        //     collide[0] = param[1] * param[2] * cos(state[2]);
        //     collide[1] = Jp + param[1] * powf(param[2], 2);
        //     collide[2] = collide[0] / collide[1];
        //     state[3] = state[3] + (1+ coefficient) * collide[2] * state[1];
        //     state[1] = -coefficient * state[1];
        //     state[0] = param[8];
        // }
        // if(state[0] <= param[7])
        // {
        //     float collide[3] = {};
        //     float coefficient = param[9];
        //     collide[0] = param[1] * param[2] * cos(state[2]);
        //     collide[1] = Jp + param[1] * powf(param[2], 2);
        //     collide[2] = collide[0] / collide[1];
        //     state[3] = state[3] + (1+coefficient) * collide[2] * state[1];
        //     state[1] = -coefficient * state[1];
        //     state[0] = param[7];
        // }

        myMPC.Set(state, SET_STATE);

        myMPC.WriteDataToFile( );
        // myMPC.WriteDataToFile( u );
    }

    time(&time_value);
    time_object = localtime( &time_value );
    while(time_object->tm_min == before_tm_min && time_object->tm_sec < 50)
    {
        sleep(8);
        time(&time_value);
        time_object = localtime( &time_value );
    }    
   return 0;
}