#include "hip/hip_runtime.h"

/*
 *  This file is part of MCMPC toolkit.
 *  
 *  MCMPC toolkit -- A toolkit for Model Predictive Control
*/
/* 
    [file]      project/example_inverted_pendulum/template.cu
    [author]    Shintaro Nakatani
    [date]      2022.6.15
*/

#include "../../include/mcmpc_toolkit.cuh"

const int OCP_SETTINGS::SIMULATION_STEPS        = 750;
const int OCP_SETTINGS::NUM_OF_PREDICTION_STEPS = 40;
const float OCP_SETTINGS::PREDICTION_INTERVAL   = 1.2f;
const float OCP_SETTINGS::CONTROL_CYCLE         = 0.02f;
const int OCP_SETTINGS::DIM_OF_STATE            = 6;
const int OCP_SETTINGS::DIM_OF_INPUT            = 1;
const int OCP_SETTINGS::DIM_OF_PARAMETER        = 8;
const int OCP_SETTINGS::DIM_OF_REFERENCE        = 6;
const int OCP_SETTINGS::DIM_OF_CONSTRAINTS      = 4;
const int OCP_SETTINGS::DIM_OF_WEIGHT_MATRIX    = 7;


/*****  *****/ 
const int CONTROLLER_PARAM::NUM_OF_SAMPLE                 = 9000;
const int CONTROLLER_PARAM::NUM_OF_ELITE_SAMPLE             = 150;
const int CONTROLLER_PARAM::NUM_OF_MONTE_CARLO_ITERATION    = 4;
const float CONTROLLER_PARAM::VARIANCE                      = 0.8f;


/***** OPTIONAL PARAMETERS *****/
const int OPTIONAL_PARAM::NUM_THREAD_PER_BLOCK      = 10;
const float OPTIONAL_PARAM::LAMBDA_GAIN             = 2e-1;

/***** PARAMETERS FOR SAMPLE-BASED NEWTON METHOD *****/
const int OPTIONAL_PARAM::NUM_OF_NEWTON_ITERATION   = 1;
const float OPTIONAL_PARAM::SBNEWTON_VARIANCE       = 0.25f;
const int OPTIONAL_PARAM::MAX_DIVISOR               = 50;

const float OPTIONAL_PARAM::COOLING_RATE            = 0.98f;

const float OPTIONAL_PARAM::BARIIER_ZETA            = 0.00001f;
const float OPTIONAL_PARAM::BARIIER_RHO             = 1e-4;
const float OPTIONAL_PARAM::BARIIER_TAU             = 1e-2;
const float OPTIONAL_PARAM::BARIIER_MAX             = 1e7;

const int OPTIONAL_PARAM::NUM_OF_GOLDEN_SEARCH_ITERATION = 4;

/***** PARAMETERS FOR MPC with CMA-ES  *****/
const int OPTIONAL_PARAM::SAMPLE_SIZE_CMA       = 40;
const int OPTIONAL_PARAM::ELITE_SAMPLE_CMA      = 7;
const float OPTIONAL_PARAM::CMA_XI              = 0.5f;
const float OPTIONAL_PARAM::LEARNING_RATE_Z     = 1.0f;
const float OPTIONAL_PARAM::LEARNING_RATE_C     = 1.0f;
const float OPTIONAL_PARAM::DAMPING_COEFFICIENT = 1.0f;
const float OPTIONAL_PARAM::PATH_UPDATE_RATE_Z  = 1.0f;
const float OPTIONAL_PARAM::PATH_UPDATE_RATE_C  = 1.0f;

/***** DYNAMIC MODEL REPRESENTING STATE TRANSITION dot{x} = "f(x,u,t,p)" *****/
__host__ __device__ void DynamicalModel(float *dx, float *x, float *u, float *param)
{
    float o[7] = { };
    o[0] = sinf(x[2] - x[4]);
    o[1] = -param[2] * o[0] * powf(x[5], 2) - (param[5] + param[6]) * x[3] + param[6] * x[5] + param[3] * param[7] * sinf(x[2]) - param[3] * cosf(x[2]) * u[0];
    o[2] = param[2] * o[0] * powf(x[3], 2) + param[6] * (x[3] - x[5]) + param[4] * param[7] * sinf(x[4]) - param[4] * cosf(x[4]) * u[0];
    o[3] = param[0] * param[1] - powf(param[2] * cosf(o[2] - o[4]), 2);
    o[4] = param[1] / o[3];
    o[5] = -param[2] * cosf(o[2] - o[4]) /o[3];
    o[6] = param[0] / o[3];
    
    dx[0] = x[1]; // dx
    dx[1] = u[0]; // ddx
    dx[2] = x[3]; // dtheta1
    dx[3] = o[4] * o[1] + o[5] * o[2]; // ddtheta1
    dx[4] = x[5]; // dtheta2
    dx[5] = o[5] * o[0] + o[6] * o[2];
    
} 

/***** FOR COMPUTING STAGE COST COST *****/
__host__ __device__ float GetStageCostTerm(float *u, float *x, float *reference, float *weight)
{
    float stage_cost = 0.0f;
    float o[2] = {};
    o[0] = sinf(x[2]/2.0f);
    o[1] = sinf(x[4]/2.0f);
    // 状態に関するステージコスト
    stage_cost += weight[0] * (x[0] - reference[0]) * (x[0] - reference[0]);
    stage_cost += weight[1] * (x[1] - reference[1]) * (x[1] - reference[1]);
    stage_cost += weight[2] * (o[0] - reference[2]) * (o[0] - reference[2]);
    stage_cost += weight[3] * (x[3] - reference[3]) * (x[3] - reference[3]);
    stage_cost += weight[4] * (o[1] - reference[4]) * (o[1] - reference[4]);
    stage_cost += weight[5] * (x[5] - reference[5]) * (x[5] - reference[5]);
    // 入力に関するステージコスト
    stage_cost += weight[6] * u[0] * u[0];
    
    stage_cost = stage_cost / 2;
    
    return stage_cost;
}

/***** FOR COMPUTING TERMINAL COST COST *****/
__host__ __device__ float GetTerminalCostTerm(float *u, float *x, float *ref, float *weight)
{
    float terminal_cost = 0.0f;

    return terminal_cost;
}

/***** FOR HANDLING INPUT CONSTRAINTS ON RANDOMLY GENERATED INPUT *****/ 
__host__ __device__ void InputSaturation(float *u, float *constraints, float zeta)
{
    u[0] = GetSaturatedInput(u[0], constraints[0], constraints[1], zeta);
}

/***** BARRIER FUNCTION FOR HANDILNG CONSTRAINTS ON STATE OR OTHERS *****/
__host__ __device__ float GetBarrierTerm(float *x, float *u, float *constraints, float rho)
{
    float log_barrier = 0.0f;
    
    log_barrier += LogBarrierConstraint(x[0], constraints[2], constraints[3], rho);

    return log_barrier;
}


