#include "hip/hip_runtime.h"
/*
 *  This file is part of MCMPC toolkit.
 *  
 *  MCMPC toolkit -- A toolkit for Model Predictive Control
*/
/* 
    [file]      src/mcmpc.cu
    [author]    Shintaro Nakatani
    [date]      2022.6.15
*/

#include "../include/cuda_check_error.cuh"
#include "../include/parallel_simulation.cuh"
#include "../include/sample_based_newton.cuh"
#include "../include/newton_fitting.cuh"
#include "../include/golden_section_search.cuh"

// コンストラクタ
sample_based_newton_method::sample_based_newton_method()
{
    // 推定入力時系列保存用配列
    CHECK( hipMallocManaged((void**)&sbnewton_input_sequences, sizeof(float) * hst_idx->input_by_horizon) );
    // ヘシアンと勾配計算用
    CHECK( hipMalloc(&hessian, sizeof(float) * hst_idx->size_of_hessian) );
    CHECK( hipMalloc(&gradient, sizeof(float) * hst_idx->input_by_horizon) );

    // QR分解に使用する行列のための配列（最小二乗法、ヘシアンの逆行列の計算で使用）
    CHECK( hipMallocManaged((void**)&coe_matrix, sizeof(float) * hst_idx->pow_hessian_elements) );
    CHECK( hipMallocManaged((void**)&tensort_x, sizeof(float) * hst_idx->sample_size_for_fitting * hst_idx->size_of_quadrtic_curve) );
    CHECK( hipMallocManaged((void**)&tensort_l, sizeof(float) * hst_idx->size_of_quadrtic_curve) );
    CHECK( hipMallocManaged((void**)&b_vector, sizeof(float) * hst_idx->sample_size_for_fitting) );

    // Set up cublas & cusolver variables
    CHECK_CUSOLVER( hipsolverDnCreate(&cusolverH), "Failed to Create cusolver handler" );
    CHECK_CUBLAS( hipblasCreate(&cublasH), "Failed to create cublas handler" );
    row_x = hst_idx->sample_size_for_fitting;
    column_x = hst_idx->size_of_quadrtic_curve;
    row_c = hst_idx->size_of_quadrtic_curve;
    row_h = hst_idx->input_by_horizon;
    alpha = 1.0f;
    beta = 0.0f;
    m_alpha = -1.0;
    CHECK( hipMalloc((void**)&cu_info, sizeof(int)) );
    CHECK( hipMalloc((void**)&qr_tau, sizeof(float) * hst_idx->size_of_quadrtic_curve) );
    CHECK( hipMalloc((void**)&hqr_tau, sizeof(float) * hst_idx->input_by_horizon) );
    uplo = HIPBLAS_FILL_MODE_LOWER;
    side = HIPBLAS_SIDE_LEFT;
    trans = HIPBLAS_OP_T;
    trans_no = HIPBLAS_OP_N;
    uplo_qr = HIPBLAS_FILL_MODE_UPPER;
    cub_diag = HIPBLAS_DIAG_NON_UNIT;

    geqrf_work_size = 0;
    ormqr_work_size = 0;
    qr_work_size = 0;
    hqr_work_size = 0;
    nrhs = 1;

    // printf("Before Generate COLDEN_SECTION_SEARCH_CLASS\n");
    // golden_section_search temp_gss;
    // gss = &temp_gss;
    // printf("After Generate COLDEN_SECTION_SEARCH_CLASS\n");
    newton_h_indices_vec = indices_dev_vec;
    block_size_qc_regression = CountBlocks(hst_idx->sample_size_for_fitting, hst_idx->thread_per_block);

    golden_section_search_flag = 0;
    mcmpc::line_search = NOT_SETTING;
}

sample_based_newton_method::~sample_based_newton_method()
{
    FreeAllCudaArrayInSBNewton();
}

void sample_based_newton_method::FreeAllCudaArrayInSBNewton()
{
    if(cusolverH) hipsolverDnDestroy(cusolverH);
    if(cublasH) hipblasDestroy(cublasH);
    CHECK( hipFree( sbnewton_input_sequences ) );
    CHECK( hipFree( hessian ) );
    CHECK( hipFree( gradient ) );
    CHECK( hipFree( coe_matrix ) );
    CHECK( hipFree( tensort_x ) );
    CHECK( hipFree( tensort_l ) );
    CHECK( hipFree( b_vector ) );
    CHECK( hipFree( qr_tau ) );
    CHECK( hipFree( hqr_tau ) );
    CHECK( hipFree( cu_info ) );
    if(solver_type == EIGEN_VALUE_DECOM) fclose(fp_fitting_accuracy);
}

void sample_based_newton_method::SetupEvaluateVariables( )
{
    time_t timeValue;
    struct tm *timeObject;
    time( &timeValue );
    timeObject = localtime( &timeValue );
    char filename[128];
    sprintf(filename, "./output/data_fitting_accuracy_%d%d_%d%d.txt", timeObject->tm_mon + 1, timeObject->tm_mday, timeObject->tm_hour,timeObject->tm_min);
    fp_fitting_accuracy = fopen(filename, "w");

    CHECK( hipMalloc((void**)&eigen_value, sizeof(float) * hst_idx->input_by_horizon) );
    CHECK( hipMalloc((void**)&diag_matrix, sizeof(float) * hst_idx->size_of_hessian) );
    CHECK( hipMalloc((void**)&orth_matrix, sizeof(float) * hst_idx->size_of_hessian) );
    CHECK( hipMalloc((void**)&regression_value, sizeof(float) * hst_idx->sample_size_for_fitting) );
    jobz = HIPSOLVER_EIG_MODE_VECTOR;
    uplo_svd = HIPBLAS_FILL_MODE_UPPER;
    thrust::device_vector<int> eigen_dev_vec_temp( hst_idx->input_by_horizon );
    eigen_hst_vec = eigen_dev_vec_temp;
    eigen_dev_vec = eigen_hst_vec;
    thrust::device_vector<float> regression_error_dev_vec_temp( hst_idx->sample_size_for_fitting, 0.0f );
    regression_error_hst_vec = regression_error_dev_vec_temp;
    regression_error_dev_vec = regression_error_hst_vec;
}

void sample_based_newton_method::ExecuteMPC(float *current_input)
{
    if(time_steps == 0 && solver_type == EIGEN_VALUE_DECOM)
    {
        SetupEvaluateVariables();
    }
    clock_t start_t, stop_t;
    start_t = clock();
    // Execute Monte Carlo Simulation
    printf("----- Start %d -th control & optimization loop -----\n", time_steps);
    mcmpc::MonteCarloSimulation();

    cost_value_mcmpc = GetCostValue(mcmpc_input_sequences, _state, _param, _ref, _cnstrnt, _weight, hst_idx);

    printf("----- Start Sample-based Newton Step Calculation -----\n");
    printf("----- Cost Value of MCMPC == %f -----\n", cost_value_mcmpc);

    float newton_variance;
    newton_variance = hst_idx->newton_search_sigma;
    
    ParallelMonteCarloSimulation<<<num_blocks, thread_per_block>>>(sample, thrust::raw_pointer_cast(sort_key_dev_vec.data()), thrust::raw_pointer_cast(indices_dev_vec.data()),
                                                                       newton_variance, _state, _param, _ref, _cnstrnt, _weight, mcmpc_input_sequences, dev_random_seed, dev_idx);
    CHECK( hipDeviceSynchronize() );
    thrust::sort_by_key(sort_key_dev_vec.begin(), sort_key_dev_vec.end(), indices_dev_vec.begin());

    GetTensortMatrices<<<block_size_qc_regression, thread_per_block>>>(tensort_x, b_vector, mcmpc_input_sequences, cost_value_mcmpc, sample, thrust::raw_pointer_cast(indices_dev_vec.data()),
                                                                      dev_idx);
    CHECK( hipDeviceSynchronize( ) );

    // Compute coe_matrix() = tensort_x^T * tensort_x 
    CHECK_CUBLAS( hipblasSgemm(cublasH, trans_no, trans, column_x, column_x, row_x, &alpha, tensort_x, column_x, tensort_x, column_x, &beta, coe_matrix, row_c), "Failed to hipblasDgemm for [coe_matrix]" );
    // CHECK( hipDeviceSynchronize());
    // sprintf(mat_name, "C");
    // printMatrix(row_c, row_c, coe_matrix, row_c, mat_name);
    // Compute tensort_l = transpose(tensort_x) * b_vector
    CHECK_CUBLAS(hipblasSgemm(cublasH, trans_no, trans, column_x, 1, row_x, &alpha, tensort_x, column_x, b_vector, 1, &beta, tensort_l, row_c), "Failed to hipblasSgemm for [tensort_l]" );
    // CHECK(hipDeviceSynchronize());

    if(time_steps == 0)
    {
        CHECK_CUSOLVER( hipsolverDnSgeqrf_bufferSize(cusolverH, row_c, row_c, coe_matrix, row_c, &geqrf_work_size), "Failed to get buffersize of [coe_matrix] (1st step)" );
        CHECK_CUSOLVER( hipsolverDnSormqr_bufferSize(cusolverH, side, trans, row_c, nrhs, row_c, coe_matrix, row_c, qr_tau, tensort_l, row_c, &ormqr_work_size), "Failed to get buffersize of [coe_matrix] (2nd step)" );
        qr_work_size = (geqrf_work_size > ormqr_work_size)? geqrf_work_size : ormqr_work_size;
        CHECK( hipMalloc((void**)&ws_qr_ops, sizeof(float) * qr_work_size) );
    }
    // Execute QR decomposition for Least Square method ==>  Q = lower triangular of coe_matrix
    CHECK_CUSOLVER( hipsolverDnSgeqrf(cusolverH, row_c, row_c, coe_matrix, row_c, qr_tau, ws_qr_ops, qr_work_size, cu_info), "Failed to compute QR factorization" );

    // Compute transpose(Q) * B for compute Ans = inv(R) * transpose(Q) * B by QR decomposition
    CHECK_CUSOLVER( hipsolverDnSormqr(cusolverH, side, trans, row_c, nrhs, row_c, coe_matrix, row_c, qr_tau, tensort_l, row_c, ws_qr_ops, qr_work_size, cu_info), "Failed to compute Q^T*B" );

    // Compute vector Ans (store Hessian elements & gradient element & constant C) = inv(R) * transpose(Q) * B
    CHECK_CUBLAS( hipblasStrsm(cublasH, side, uplo_qr, trans_no, cub_diag, row_c, nrhs, &alpha, coe_matrix, row_c, tensort_l, row_c), "Failed to compute X = R^-1Q^T*B" );

    GetHessinaAndGradient<<<hst_idx->input_by_horizon, hst_idx->input_by_horizon>>>(hessian, gradient, tensort_l, dev_idx);
    CHECK( hipDeviceSynchronize( ) );

    switch(solver_type){
        case EIGEN_VALUE_DECOM:
            if(time_steps == 0)
            {
                CHECK_CUSOLVER( hipsolverDnSgeqrf_bufferSize(cusolverH, row_h, row_h, hessian, row_h, &geqrf_work_size), "Failed to get buffersize of [Hessian]<=1" );
                CHECK_CUSOLVER( hipsolverDnSormqr_bufferSize(cusolverH, side, trans, row_h, nrhs, row_h, hessian, row_h, hqr_tau, gradient, row_h, &ormqr_work_size), "Failed to get buffersize for QR decom [2]" );
                hqr_work_size = (geqrf_work_size > ormqr_work_size)? geqrf_work_size : ormqr_work_size;
                CHECK( hipMalloc((void**)&ws_hqr_ops, sizeof(float) * hqr_work_size) );
                CHECK_CUSOLVER( hipsolverDnSsyevd_bufferSize(cusolverH,jobz, uplo, row_h, hessian, row_h, eigen_value, &hsvd_work_size), "Faile to get buffersize of [Hessian]<=>1");
                CHECK(hipMalloc((void**)&ws_hsvd_ops, sizeof(float) * hsvd_work_size) );
            }
            CHECK_CUSOLVER( hipsolverDnSsyevd(cusolverH, jobz, uplo, row_h, hessian, row_h, eigen_value, ws_hsvd_ops, hsvd_work_size, cu_info), "Failed to decompose singular value of Hesian" );
            GetEigenValueInfo<<<hst_idx->input_by_horizon, hst_idx->input_by_horizon>>>(thrust::raw_pointer_cast(eigen_dev_vec.data()), diag_matrix, eigen_value, dev_idx);
            CHECK( hipDeviceSynchronize() );
            thrust::inclusive_scan(eigen_dev_vec.begin(), eigen_dev_vec.end(), eigen_dev_vec.begin());
            eigen_hst_vec = eigen_dev_vec;
            printf("num of negative eigen values = %d\n", eigen_hst_vec[hst_idx->input_by_horizon - 1]);

            CHECK_CUBLAS( hipblasSgemm(cublasH, trans_no, trans_no, row_h, row_h, row_h, &alpha, hessian, row_h, diag_matrix, row_h, &beta, orth_matrix, row_h), "Failed to compute  inverse matrix 1st operation" );
            CHECK_CUBLAS( hipblasSgemm(cublasH, trans_no, trans, row_h, row_h, row_h, &alpha, orth_matrix, row_h, hessian, row_h, &beta, diag_matrix, row_h), "Failed to compute inv(2.0*Hessian)" );
            CHECK_CUSOLVER( hipsolverDnSgeqrf(cusolverH, row_h, row_h, diag_matrix, row_h, hqr_tau, ws_hqr_ops, hqr_work_size, cu_info),"Failed to compute QR factorization of Hessain" );
            CHECK_CUSOLVER( hipsolverDnSormqr(cusolverH, side, trans, row_h, nrhs, row_h, diag_matrix, row_h, hqr_tau, gradient, row_h, ws_hqr_ops, hqr_work_size, cu_info), "Failed to compute Q^T*B of Hessian" );
            CHECK_CUBLAS( hipblasStrsm(cublasH, side, uplo_svd, trans_no, cub_diag, row_h, nrhs, &m_alpha, diag_matrix, row_h, gradient, row_h), "Failed to compute X = R^-1Q^T*B" );
            ComputeNewtonStep<<<hst_idx->input_by_horizon, 1>>>(sbnewton_input_sequences, mcmpc_input_sequences, gradient);
            CHECK( hipDeviceSynchronize() );

            // 回帰超２次超曲面における予測評価値を元のサンプルの評価値の昇順にregResultsVecに格納して評価する
            ResetTensortMatrices<<<block_size_qc_regression, thread_per_block>>>(tensort_x, sample, thrust::raw_pointer_cast(indices_dev_vec.data()), dev_idx);
            CHECK( hipDeviceSynchronize() );
            CHECK_CUBLAS( hipblasSgemm(cublasH, trans,  trans_no, column_x, 1, column_x, &alpha, tensort_x, column_x, tensort_l, column_x, &beta, regression_value, column_x), "Failed to compute predictive cost value !" );

#ifdef MEAN_ABSOLUTE_ERROR
            GetMeanAbsoluteError<<<block_size_qc_regression, thread_per_block>>>(thrust::raw_pointer_cast(regression_error_dev_vec.data()), regression_value, sample, thrust::raw_pointer_cast(indices_dev_vec.data()), dev_idx);
            CHECK( hipDeviceSynchronize() );
            thrust::inclusive_scan(regression_error_dev_vec.begin(), regression_error_dev_vec.end(), regression_error_dev_vec.begin());
            regression_error_hst_vec = regression_error_dev_vec;
            regression_accuracy = regression_error_hst_vec[hst_idx->sample_size_for_fitting - 1] / hst_idx->sample_size_for_fitting;
#else
            GetMeanSquareError<<<block_size_qc_regression, thread_per_block>>>(thrust::raw_pointer_cast(regression_error_dev_vec.data()), regression_value, sample, thrust::raw_pointer_cast(indices_dev_vec.data()), dev_idx);
            CHECK( hipDeviceSynchronize() );
            thrust::inclusive_scan(regression_error_dev_vec.begin(), regression_error_dev_vec.end(), regression_error_dev_vec.begin());
            regression_error_hst_vec = regression_error_dev_vec;
            regression_accuracy = sqrt( regression_error_hst_vec[hst_idx->sample_size_for_fitting - 1] / hst_idx->sample_size_for_fitting );
#endif
            break;
        case QR_DECOM:
            if(time_steps == 0)
            {
                CHECK_CUSOLVER( hipsolverDnSgeqrf_bufferSize(cusolverH, row_h, row_h, hessian, row_h, &geqrf_work_size), "Failed to get buffersize of [Hessian] (1st step)" );
                CHECK_CUSOLVER( hipsolverDnSormqr_bufferSize(cusolverH, side, trans, row_h, nrhs, row_h, hessian, row_h, hqr_tau, gradient, row_h, &ormqr_work_size), "Failed to get buffersize for QR decom [2]" );
                hqr_work_size = (geqrf_work_size > ormqr_work_size)? geqrf_work_size : ormqr_work_size;
                CHECK( hipMalloc(&ws_hqr_ops, sizeof(float) * hqr_work_size) );
            }
            CHECK_CUSOLVER( hipsolverDnSgeqrf(cusolverH, row_h, row_h, hessian, row_h, hqr_tau, ws_hqr_ops, hqr_work_size, cu_info),"Failed to compute QR factorization of Hessain" );
            CHECK_CUSOLVER( hipsolverDnSormqr(cusolverH, side, trans, row_h, nrhs, row_h, hessian, row_h, hqr_tau, gradient, row_h, ws_hqr_ops, hqr_work_size, cu_info), "Failed to compute Q^T*B of Hessian" )
            CHECK_CUBLAS( hipblasStrsm(cublasH, side, uplo_qr, trans_no, cub_diag, row_h, nrhs, &m_alpha, hessian, row_h, gradient, row_h), "Failed to compute X = R^-1Q^T*B" );
            ComputeNewtonStep<<<hst_idx->input_by_horizon, 1>>>(sbnewton_input_sequences, mcmpc_input_sequences, gradient);
            CHECK( hipDeviceSynchronize() );
            break;
        default:
            break;
    }
    // if(time_steps == 0)
    // {
    //     CHECK_CUSOLVER( hipsolverDnSgeqrf_bufferSize(cusolverH, row_h, row_h, hessian, row_h, &geqrf_work_size), "Failed to get buffersize of [Hessian] (1st step)" );
    //     CHECK_CUSOLVER( hipsolverDnSormqr_bufferSize(cusolverH, side, trans, row_h, nrhs, row_h, hessian, row_h, hqr_tau, gradient, row_h, &ormqr_work_size), "Failed to get buffersize for QR decom [2]" );
    //     hqr_work_size = (geqrf_work_size > ormqr_work_size)? geqrf_work_size : ormqr_work_size;
    //     CHECK( hipMalloc(&ws_hqr_ops, sizeof(float) * hqr_work_size) );
    // }
    // // Execute QR decomposition to get inv(Hessian)  ==>  Q = lower triangular of coe_matrix
    // CHECK_CUSOLVER( hipsolverDnSgeqrf(cusolverH, row_h, row_h, hessian, row_h, hqr_tau, ws_hqr_ops, hqr_work_size, cu_info),"Failed to compute QR factorization of Hessain" );

    // // Compute transpose(Q) * B for compute Ans (inv(Hessian) * Gradient) = inv(R) * transpose(Q) * B by QR decomposition
    // CHECK_CUSOLVER( hipsolverDnSormqr(cusolverH, side, trans, row_h, nrhs, row_h, hessian, row_h, hqr_tau, gradient, row_h, ws_hqr_ops, hqr_work_size, cu_info), "Failed to compute Q^T*B of Hessian" )

    // // Compute estimated input sequences = inv(R) * transpose(Q) * B
    // CHECK_CUBLAS( hipblasStrsm(cublasH, side, uplo_qr, trans_no, cub_diag, row_h, nrhs, &m_alpha, hessian, row_h, gradient, row_h), "Failed to compute X = R^-1Q^T*B" );

    // ComputeNewtonStep<<<hst_idx->input_by_horizon, 1>>>(sbnewton_input_sequences, mcmpc_input_sequences, gradient);
    // CHECK( hipDeviceSynchronize() );

    // cost_value_newton = GetCostValue(sbnewton_input_sequences, _state, _param, _ref, _cnstrnt, _weight, hst_idx);

    GetCostValueNewton(cost_value_newton, check_violate_constraint, sbnewton_input_sequences, _state, _param, _ref, _cnstrnt, _weight, hst_idx);

    printf("----- End Sample-based Newton Step Calculation -----\n");
    printf("----- Cost value of Sample-based Newton method == %f ----\n",cost_value_newton);
    
    cost_value_newton_after_gss = cost_value_newton;
    if(line_search == GOLDEN_SECTION)
    {
        golden_section_search::ExeGoldenSectionSearch( cost_value_newton_after_gss, cost_value_mcmpc, sbnewton_input_sequences, mcmpc_input_sequences, sample, 
                                                    thrust::raw_pointer_cast(newton_h_indices_vec.data()), _state, _param, _ref, _cnstrnt, _weight);
        printf("----- Cost value of Sample-based Newton method after golden search == %f ----\n",cost_value_newton_after_gss);
    }
    
    
    // printf("----- Cost value of Sample-based Newton method after golden search == %f ----\n",cost_value_newton);
    stop_t = clock();
    all_time = stop_t - start_t;
    printf("*** Computation time of Sample-based Newton method = [%f] ***\n", all_time / CLOCKS_PER_SEC);
    SelectOptimalSolution( current_input );
    time_steps++;
}

void sample_based_newton_method::SelectOptimalSolution( float *current_input )
{
    if(cost_value_newton <= cost_value_mcmpc || cost_value_newton_after_gss <= cost_value_mcmpc){
        // if(cost_value_newton_after_gss == cost_value_mcmpc) cost_value_newton_after_gss -= 1e-4;
        golden_section_search_flag = 1;
        SetInputSequences<<<hst_idx->input_by_horizon, 1>>>(mcmpc_input_sequences, sbnewton_input_sequences);
        CHECK( hipDeviceSynchronize() );
        for(int i = 0; i < hst_idx->dim_of_input; i++)
        {
            current_input[i] = mcmpc_input_sequences[i];
        }
        printf("*** Newton method superior than MCMPC ***\n");
        cumsum_cost += cost_value_newton_after_gss / hst_idx->horizon;
    }else{
        golden_section_search_flag = 0;
        for(int i = 0; i < hst_idx->dim_of_input; i++)
        {
            current_input[i] = mcmpc_input_sequences[i];
        }
        printf("*** SB-Newton method inferior than MCMPC ***\n");
        cumsum_cost += cost_value_mcmpc / hst_idx->horizon;
    }
}

void sample_based_newton_method::WriteDataToFile( )
{
    float current_time = time_steps * hst_idx->control_cycle;
    for(int i = 0; i < hst_idx->dim_of_state; i++)
    {
        if(i == 0)
        {
            fprintf(fp_state, "%f %f ", current_time, _state[i]);
        }else if(i == hst_idx->dim_of_state - 1){
            fprintf(fp_state, "%f\n", _state[i]);
        }else{
            fprintf(fp_state, "%f ", _state[i]);
        }
    }

    if(line_search == GOLDEN_SECTION)
    {
        if(cost_value_mcmpc < cost_value_newton_after_gss) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_mcmpc, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
        if(cost_value_newton_after_gss <= cost_value_mcmpc) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_newton_after_gss, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
    }else{
        if(cost_value_mcmpc < cost_value_newton_after_gss) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_mcmpc, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
        if(cost_value_newton_after_gss <= cost_value_mcmpc) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_newton_after_gss, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
    }

    for(int i = 0; i < hst_idx->dim_of_input; i++)
    {
        if(i == 0)
        {
            if(!(hst_idx->dim_of_input - 1) == 0)
            {
                fprintf(fp_input, "%f %f ", current_time, mcmpc_input_sequences[i]);
            }else{
                fprintf(fp_input, "%f %f\n", current_time, mcmpc_input_sequences[i]);
            }
        }else if(i == hst_idx->dim_of_input - 1){
            fprintf(fp_input, "%f\n", mcmpc_input_sequences[i]);
        }else{
            fprintf(fp_input, "%f ", mcmpc_input_sequences[i]);
        }
    }

    if(solver_type == EIGEN_VALUE_DECOM)
    {
        int regression_id = floor(hst_idx->sample_size_for_fitting / 10) - 1;
        float den_r = regression_error_hst_vec[hst_idx->sample_size_for_fitting - 1];
        if(0 < eigen_hst_vec[hst_idx->input_by_horizon - 1]){
            fprintf(fp_fitting_accuracy, "%f %d %f ", current_time, 1, regression_accuracy);
        }else{
            fprintf(fp_fitting_accuracy, "%f %d %f ", current_time, 0, regression_accuracy);
        }
        for(int i = 0; i < 9; i++)
        {
            if(i < 8) fprintf(fp_fitting_accuracy, "%f ", regression_error_hst_vec[(i+1)*regression_id] / den_r);
            if(i == 8) fprintf(fp_fitting_accuracy, "%f\n", regression_error_hst_vec[(i+1)*regression_id] / den_r);
        }
    }
}

void sample_based_newton_method::WriteDataToFile(float *_input)
{
    float current_time = time_steps * hst_idx->control_cycle;
    for(int i = 0; i < hst_idx->dim_of_state; i++)
    {
        if(i == 0)
        {
            fprintf(fp_state, "%f %f ", current_time, _state[i]);
        }else if(i == hst_idx->dim_of_state - 1){
            fprintf(fp_state, "%f\n", _state[i]);
        }else{
            fprintf(fp_state, "%f ", _state[i]);
        }
    }

    if(line_search == GOLDEN_SECTION)
    {
        if(cost_value_mcmpc < cost_value_newton_after_gss) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_mcmpc, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
        if(cost_value_newton_after_gss <= cost_value_mcmpc) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_newton_after_gss, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
    }else{
        if(cost_value_mcmpc < cost_value_newton_after_gss) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_mcmpc, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
        if(cost_value_newton_after_gss <= cost_value_mcmpc) fprintf(fp_cost, "%f %f %f %f %f %f %d %f\n", current_time, cost_value_newton_after_gss, cumsum_cost, cost_value_mcmpc, cost_value_newton, cost_value_newton_after_gss, golden_section_search_flag, all_time / CLOCKS_PER_SEC);
    }
    

    for(int i = 0; i < hst_idx->dim_of_input; i++)
    {
        if(i == 0)
        {
            if(!(hst_idx->dim_of_input - 1) == 0)
            {
                fprintf(fp_input, "%f %f ", current_time, _input[i]);
            }else{
                fprintf(fp_input, "%f %f\n", current_time, _input[i]);
            }
        }else if(i == hst_idx->dim_of_input - 1){
            fprintf(fp_input, "%f\n", _input[i]);
        }else{
            fprintf(fp_input, "%f ", _input[i]);
        }
    }

    if(solver_type == EIGEN_VALUE_DECOM)
    {
        int regression_id = floor(hst_idx->sample_size_for_fitting / 10) - 1;
        float den_r = regression_error_hst_vec[hst_idx->sample_size_for_fitting - 1];
        if(0 < eigen_hst_vec[hst_idx->input_by_horizon - 1]){
            fprintf(fp_fitting_accuracy, "%f %d %f ", current_time, 1, regression_accuracy);
        }else{
            fprintf(fp_fitting_accuracy, "%f %d %f ", current_time, 0, regression_accuracy);
        }
        for(int i = 0; i < 9; i++)
        {
            if(i < 8) fprintf(fp_fitting_accuracy, "%f ", regression_error_hst_vec[(i+1)*regression_id] / den_r);
            if(i == 8) fprintf(fp_fitting_accuracy, "%f\n", regression_error_hst_vec[(i+1)*regression_id] / den_r);
        }
    }
}

// void sample_based_newton_method::printMatrix(int m, int n, float *A, int lda, const char* name)
// {
//     FILE *temp_mat_file;
//     char f_m_n[128];
//     sprintf(f_m_n, "./output/matrix_%s_time_step_%d.txt", name, time_steps);
//     temp_mat_file = fopen(f_m_n, "w");
//     for(int row = 0 ; row < m ; row++){
//         for(int col = 0 ; col < n ; col++){
//             double Areg = A[row + col*lda];
//             printf("%s(%d,%d) = %lf\n", name, row+1, col+1, Areg);
//             //printf("%s[%d] = %lf\n", name, row + col*lda, Areg);
//             if(col == n-1) fprintf(temp_mat_file, "%lf\n", Areg);
//             if(col < n -1) fprintf(temp_mat_file, "%lf ", Areg);
//         }
//     }
//     // sleep(2);
//     fclose(temp_mat_file);
// }